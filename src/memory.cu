int InitMemStructs(MemStruct * HostMem, MemStruct * DeviceMem, SimulationStruct * sim) {
  int rz_size, ra_size;
  rz_size = sim -> det.nr * sim -> det.nz;
  ra_size = sim -> det.nr * sim -> det.na;

  hipMalloc((void ** ) & DeviceMem -> p, NUM_THREADS * sizeof(PhotonStruct));

  HostMem -> A_rz = (unsigned long long * ) malloc(rz_size * sizeof(unsigned long long));

  if (HostMem -> A_rz == NULL) {
    printf("Error allocating HostMem->A_rz");
    exit(1);
  }

  hipMalloc((void ** ) & DeviceMem -> A_rz, rz_size * sizeof(unsigned long long));
  hipMemset(DeviceMem -> A_rz, 0, rz_size * sizeof(unsigned long long));

  HostMem -> Rd_ra = (unsigned long long * ) malloc(ra_size * sizeof(unsigned long long));

  if (HostMem -> Rd_ra == NULL) {
    printf("Error allocating HostMem->Rd_ra");
    exit(1);
  }

  hipMalloc((void ** ) & DeviceMem -> Rd_ra, ra_size * sizeof(unsigned long long));
  hipMemset(DeviceMem -> Rd_ra, 0, ra_size * sizeof(unsigned long long));

  HostMem -> Tt_ra = (unsigned long long * ) malloc(ra_size * sizeof(unsigned long long));

  if (HostMem -> Tt_ra == NULL) {
    printf("Error allocating HostMem->Tt_ra");
    exit(1);
  }

  hipMalloc((void ** ) & DeviceMem -> Tt_ra, ra_size * sizeof(unsigned long long));
  hipMemset(DeviceMem -> Tt_ra, 0, ra_size * sizeof(unsigned long long));

  hipMalloc((void ** ) & DeviceMem -> x, NUM_THREADS * sizeof(unsigned long long));
  hipMemcpy(DeviceMem -> x, HostMem -> x, NUM_THREADS * sizeof(unsigned long long), hipMemcpyHostToDevice);
  hipMalloc((void ** ) & DeviceMem -> a, NUM_THREADS * sizeof(unsigned int));
  hipMemcpy(DeviceMem -> a, HostMem -> a, NUM_THREADS * sizeof(unsigned int), hipMemcpyHostToDevice);

  HostMem -> thread_active = (unsigned int * ) malloc(NUM_THREADS * sizeof(unsigned int));

  if (HostMem -> thread_active == NULL) {
    printf("Error allocating HostMem->thread_active");
    exit(1);
  }

  for (int i = 0; i < NUM_THREADS; i++) HostMem -> thread_active[i] = 1u;

  hipMalloc((void ** ) & DeviceMem -> thread_active, NUM_THREADS * sizeof(unsigned int));
  hipMemcpy(DeviceMem -> thread_active, HostMem -> thread_active, NUM_THREADS * sizeof(unsigned int), hipMemcpyHostToDevice);

  HostMem -> num_terminated_photons = (unsigned int * ) malloc(sizeof(unsigned int));

  if (HostMem -> num_terminated_photons == NULL) {
    printf("Error allocating HostMem->num_terminated_photons");
    exit(1);
  }

  * HostMem -> num_terminated_photons = 0;

  hipMalloc((void ** ) & DeviceMem -> num_terminated_photons, sizeof(unsigned int));
  hipMemcpy(DeviceMem -> num_terminated_photons, HostMem -> num_terminated_photons, sizeof(unsigned int), hipMemcpyHostToDevice);

  return 1;
}

int CopyDeviceToHostMem(MemStruct * HostMem, MemStruct * DeviceMem, SimulationStruct * sim) {
  int rz_size = sim -> det.nr * sim -> det.nz;
  int ra_size = sim -> det.nr * sim -> det.na;

  hipMemcpy(HostMem -> A_rz, DeviceMem -> A_rz, rz_size * sizeof(unsigned long long), hipMemcpyDeviceToHost);
  hipMemcpy(HostMem -> Rd_ra, DeviceMem -> Rd_ra, ra_size * sizeof(unsigned long long), hipMemcpyDeviceToHost);
  hipMemcpy(HostMem -> Tt_ra, DeviceMem -> Tt_ra, ra_size * sizeof(unsigned long long), hipMemcpyDeviceToHost);
  hipMemcpy(HostMem -> x, DeviceMem -> x, NUM_THREADS * sizeof(unsigned long long), hipMemcpyDeviceToHost);

  return 0;
}

int InitDCMem(SimulationStruct * sim) {
  hipMemcpyToSymbol(HIP_SYMBOL(det_dc), & (sim -> det), sizeof(DetStruct));
  hipMemcpyToSymbol(HIP_SYMBOL(n_layers_dc), & (sim -> n_layers), sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(start_weight_dc), & (sim -> start_weight), sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(layers_dc), sim -> layers, (sim -> n_layers + 2) * sizeof(LayerStruct));
  hipMemcpyToSymbol(HIP_SYMBOL(num_photons_dc), & (sim -> number_of_photons), sizeof(unsigned int));

  return 0;
}