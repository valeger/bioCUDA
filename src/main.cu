#include "hip/hip_runtime.h"
#include <float.h>
#include <stdio.h>

#include "structs.h"

__device__ __constant__ unsigned int num_photons_dc[1];
__device__ __constant__ unsigned int n_layers_dc[1];
__device__ __constant__ unsigned int start_weight_dc[1];
__device__ __constant__ LayerStruct layers_dc[MAX_LAYERS];
__device__ __constant__ DetStruct det_dc[1];

#include "memory.cu"
#include "io.cu"
#include "randomgen.cu"
#include "transport.cu"


void DoOneSimulation(SimulationStruct * simulation, unsigned long long * x, unsigned int * a) {
  MemStruct DeviceMem;
  MemStruct HostMem;
  unsigned int threads_active_total = 1;
  unsigned int i, ii;

  hipError_t cudastat;
  clock_t time1, time2;

  time1 = clock();

  HostMem.x = x;
  HostMem.a = a;

  InitMemStructs( & HostMem, & DeviceMem, simulation);
  InitDCMem(simulation);

  dim3 dimBlock(NUM_THREADS_PER_BLOCK);
  dim3 dimGrid(NUM_BLOCKS);

  LaunchPhoton_Global <<< dimGrid, dimBlock >>> (DeviceMem);
  hipDeviceSynchronize(); 
  cudastat = hipGetLastError();
  if (cudastat) printf("Error code=%i, %s.\n", cudastat, hipGetErrorString(cudastat));

  printf("ignoreAdetection = %d\n\n", simulation -> ignoreAdetection);

  i = 0;
  while (threads_active_total > 0) {
    i++;
    if (simulation -> ignoreAdetection == 1) {
      MCd < 1 > <<< dimGrid, dimBlock >>> (DeviceMem);
    } else {
      MCd < 0 > <<< dimGrid, dimBlock >>> (DeviceMem);
    }
    hipDeviceSynchronize(); 
    cudastat = hipGetLastError();
    if (cudastat) printf("Error code=%i, %s.\n", cudastat, hipGetErrorString(cudastat));

    hipMemcpy(HostMem.thread_active, DeviceMem.thread_active, NUM_THREADS * sizeof(unsigned int), hipMemcpyDeviceToHost);
    threads_active_total = 0;
    for (ii = 0; ii < NUM_THREADS; ii++) threads_active_total += HostMem.thread_active[ii];

    hipMemcpy(HostMem.num_terminated_photons, DeviceMem.num_terminated_photons, sizeof(unsigned int), hipMemcpyDeviceToHost);

    printf("Run %u, Number of photons terminated %u, Threads active %u\n", i, * HostMem.num_terminated_photons, threads_active_total);
  }
  printf("Simulation done!\n");

  CopyDeviceToHostMem( & HostMem, & DeviceMem, simulation);

  time2 = clock();

  printf("Simulation time: %.2f sec\n", (double)(time2 - time1) / CLOCKS_PER_SEC);

  Write_Simulation_Results( & HostMem, simulation, time2 - time1);

  FreeMemStructs( & HostMem, & DeviceMem);
}

int main(int argc, char * argv[]) {
  int i;
  SimulationStruct * simulations;
  int n_simulations;
  unsigned long long seed = (unsigned long long) time(NULL);
  int ignoreAdetection = 0;
  char * filename;

  if (argc < 2) {
    printf("Not enough input arguments!\n");
    return 1;
  } else {
    filename = argv[1];
  }

  if (interpret_arg(argc, argv, & seed, & ignoreAdetection)) return 1;

  n_simulations = read_simulation_data(filename, & simulations, ignoreAdetection);

  if (n_simulations == 0) {
    printf("Something wrong with read_simulation_data!\n");
    return 1;
  } else {
    printf("Read %d simulations\n", n_simulations);
  }

  unsigned long long x[NUM_THREADS];
  unsigned int a[NUM_THREADS];

  if (init_RNG(x, a, NUM_THREADS, "trueprimes.txt", seed)) return 1;

  for (i = 0; i < n_simulations; i++) {
    DoOneSimulation( & simulations[i], x, a);
  }

  FreeSimulationStruct(simulations, n_simulations);

  return 0;
}